#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thread>
#include <atomic>
#include <random>
#include <chrono>
#include <cassert>
#include <unordered_set>
#include <mutex>

#include "allocator.h"

// lets make better error codes

enum class AllocationError {
    SUCCESS = 0,
    NULL_POINTER,
    INVALID_SLAB_INDEX,
    SLAB_NOT_BOUND,
    OFFSET_TOO_SMALL,
    MISALIGNED_OBJECT,
    OBJECT_INDEX_OUT_OF_RANGE,
    TRACKING_INDEX_OUT_OF_RANGE,
    CAS_FAILURE_ALREADY_ALLOCATED,
    CAS_FAILURE_DOUBLE_FREE,
    ALLOCATOR_FAILED,
    TRACKER_RECORDING_FAILED
};

const char* errorToString(AllocationError error) {
    switch (error) {
        case AllocationError::SUCCESS: return "SUCCESS";
        case AllocationError::NULL_POINTER: return "NULL_POINTER";
        case AllocationError::INVALID_SLAB_INDEX: return "INVALID_SLAB_INDEX";
        case AllocationError::SLAB_NOT_BOUND: return "SLAB_NOT_BOUND";
        case AllocationError::OFFSET_TOO_SMALL: return "OFFSET_TOO_SMALL";
        case AllocationError::MISALIGNED_OBJECT: return "MISALIGNED_OBJECT";
        case AllocationError::OBJECT_INDEX_OUT_OF_RANGE: return "OBJECT_INDEX_OUT_OF_RANGE";
        case AllocationError::TRACKING_INDEX_OUT_OF_RANGE: return "TRACKING_INDEX_OUT_OF_RANGE";
        case AllocationError::CAS_FAILURE_ALREADY_ALLOCATED: return "CAS_FAILURE_ALREADY_ALLOCATED";
        case AllocationError::CAS_FAILURE_DOUBLE_FREE: return "CAS_FAILURE_DOUBLE_FREE";
        case AllocationError::ALLOCATOR_FAILED: return "ALLOCATOR_FAILED";
        case AllocationError::TRACKER_RECORDING_FAILED: return "TRACKER_RECORDING_FAILED";
        default: return "UNKNOWN_ERROR";
    }
}

// global tracker for allocations/frees using an atomic bitmask
class ParallelTracker {
public:
    static constexpr size_t PER_SLAB_STRIDE = 1024;
    static constexpr size_t MAX_TRACKED_OBJECTS = 256 * PER_SLAB_STRIDE;
private:
    // each entry: [31:16] = size, [15:0] = thread_id, 0 = free
    // std::atomic<uint32_t> trackingArena[MAX_TRACKED_OBJECTS];
    std::unique_ptr<std::atomic<uint32_t>[]> trackingArena;
    std::atomic<size_t> totalAllocations{0};
    std::atomic<size_t> totalFrees{0};
    std::atomic<size_t> totalFailures{0};

    mutable std::mutex debugMutex;
    
public:

    ParallelTracker():trackingArena(new std::atomic<uint32_t>[MAX_TRACKED_OBJECTS]) {
        for (size_t i = 0; i < MAX_TRACKED_OBJECTS; ++i) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }

    // figure out arena index for a pointer
    std::pair<size_t, AllocationError> getIndexForPointer(void* ptr, TestSlabArena& arena) {
        if (!ptr) {
            return {MAX_TRACKED_OBJECTS, AllocationError::NULL_POINTER};
        }

        // which slab?
        auto slabIndex = arena.slabIndexFor(ptr);
        if (slabIndex >= TestSlabArena::SLAB_COUNT) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: ptr=" << ptr << " -> slabIndex=" << slabIndex 
                        << " (>= SLAB_COUNT=" << TestSlabArena::SLAB_COUNT << ")" << std::endl;
            
            // LSF analysis - show pointer details
            std::cout << "    LSF: ptr=0x" << std::hex << reinterpret_cast<uintptr_t>(ptr) 
                        << std::dec << " (LSF=" << (reinterpret_cast<uintptr_t>(ptr) & 0xF) << ")" << std::endl;
            std::cout << "    Checking all slabs for potential matches:" << std::endl;
            for (size_t i = 0; i < TestSlabArena::SLAB_COUNT; i++) {
                auto& slab = arena.slabAt(i);
                char* slabStart = reinterpret_cast<char*>(&slab);
                char* slabEnd = slabStart + TestSlabArena::slabType::SIZE;
                char* ptrChar = static_cast<char*>(ptr);
                
                if (ptrChar >= slabStart && ptrChar < slabEnd) {
                    std::cout << "      Found ptr in slab " << i << " (range: " 
                                << static_cast<void*>(slabStart) << " - " 
                                << static_cast<void*>(slabEnd) << ")" << std::endl;
                }
            }
            
            return {MAX_TRACKED_OBJECTS, AllocationError::INVALID_SLAB_INDEX};
        }

        // read proxy + slab
        auto& proxy = arena.proxyAt(slabIndex).data;
        if (proxy.getSize() == 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: slabIndex=" << slabIndex << " not bound (size=0)" << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::SLAB_NOT_BOUND};
        }

        auto& slab = arena.slabAt(slabIndex);

        // re-compute object index like proxy.free()
        char* p   = static_cast<char*>(ptr);
        char* sb  = reinterpret_cast<char*>(&slab);

        size_t objectSize   = proxy.getSize();
        size_t maxObjCount  = proxy.slabObjCount(objectSize);
        if (maxObjCount == 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: slabIndex=" << slabIndex << " objectSize=" << objectSize 
                        << " -> maxObjCount=0" << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::OBJECT_INDEX_OUT_OF_RANGE};
        }

        // number of mask elements and their byte size
        using allocMaskElem = typename TestSlabArena::slabProxyType::allocMaskElem;
        constexpr size_t ELEM_BITS = sizeof(allocMaskElem) * 8;

        size_t maskCount = (maxObjCount + ELEM_BITS - 1) / ELEM_BITS;
        size_t maskBytes = maskCount * sizeof(allocMaskElem);

        size_t firstObjOffset = maskBytes;

        size_t byteOffset = static_cast<size_t>(p - sb);
        if (byteOffset < firstObjOffset) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: byteOffset=" << byteOffset << " < firstObjOffset=" 
                        << firstObjOffset << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::OFFSET_TOO_SMALL};
        }

        size_t objOffset = byteOffset - firstObjOffset;
        if (objOffset % objectSize != 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: objOffset=" << objOffset << " % objectSize=" 
                        << objectSize << " = " << (objOffset % objectSize) << " (not aligned)" << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::MISALIGNED_OBJECT};
        }

        size_t objIndex = objOffset / objectSize;
        if (objIndex >= maxObjCount) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: objIndex=" << objIndex << " >= maxObjCount=" 
                        << maxObjCount << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::OBJECT_INDEX_OUT_OF_RANGE};
        }

        // flat index with fixed stride
        size_t idx = static_cast<size_t>(slabIndex) * PER_SLAB_STRIDE + objIndex;
        if (idx >= MAX_TRACKED_OBJECTS) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: computed index=" << idx << " >= MAX_TRACKED_OBJECTS=" 
                        << MAX_TRACKED_OBJECTS << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::TRACKING_INDEX_OUT_OF_RANGE};
        }

        return {idx, AllocationError::SUCCESS};
    } // end of getInd
    
    // wrapper
    size_t getIndexForPtr(void* ptr, TestSlabArena& arena) {
        auto result = getIndexForPointer(ptr, arena);
        return result.first;
    }
    
    // log an allocation with CAS
    bool recordAllocation(void* ptr, size_t size, uint16_t threadId, TestSlabArena& arena) {
        auto result = getIndexForPointer(ptr, arena);
        size_t index = result.first;
        AllocationError error = result.second;
        
        if (error != AllocationError::SUCCESS) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "ALLOC FAIL: Thread=" << threadId << " ptr=" << ptr 
                      << " sz=" << size << " ind=" << index 
                      << " error=" << errorToString(error) << std::endl;
            totalFailures.fetch_add(1, std::memory_order_relaxed);
            return false;
        }
        
        uint32_t newValue = (static_cast<uint32_t>(size & 0xFFFF) << 16) | (threadId & 0xFFFF);
        uint32_t expected = 0;
        
        if (trackingArena[index].compare_exchange_strong(expected, newValue, std::memory_order_acq_rel)) {
            totalAllocations.fetch_add(1, std::memory_order_relaxed);
            return true;
        }
        
        // CAS failed - slot was already occupied
        {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "ALLOC CAS FAIL: Thread=" << threadId << " ptr=" << ptr 
                      << " sz=" << size << " ind=" << index 
                      << " expected=0 actual=" << expected 
                      << " (thread=" << (expected & 0xFFFF) 
                      << " size=" << ((expected >> 16) & 0xFFFF) << ")" << std::endl;
        }
        
        totalFailures.fetch_add(1, std::memory_order_relaxed);
        return false;
    } // uped
    
    // log a free with CAS
    bool recordFree(void* ptr, uint16_t threadId, TestSlabArena& arena) {
        auto result = getIndexForPointer(ptr, arena);
        size_t index = result.first;
        AllocationError error = result.second;
        
        if (error != AllocationError::SUCCESS) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "FREE FAIL: Thread=" << threadId << " ptr=" << ptr 
                      << " ind=" << index << " error=" << errorToString(error) << std::endl;
            return false; // Don't count as total failure since this is just tracking
        }

        uint32_t curr = trackingArena[index].load(std::memory_order_acquire);
        while (curr != 0) {
            // clear to 0; CAS updates `cur` on failure
            if (trackingArena[index].compare_exchange_weak(curr, 0, std::memory_order_acq_rel)) {
                totalFrees.fetch_add(1, std::memory_order_relaxed);
                return true;
            }
        }
        
        // Was already 0 - possible double free or race
        if (curr == 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "FREE CAS WARN: Thread=" << threadId << " ptr=" << ptr 
                      << " ind=" << index << " already freed (double-free or race)" << std::endl;
        }
        
        // Treat as success since the slot is clear
        return true;
    }
 
    
    // dump stats
    void getStats(size_t& allocs, size_t& frees, size_t& failures, size_t& leaks) {
        allocs = totalAllocations.load(std::memory_order_relaxed);
        frees = totalFrees.load(std::memory_order_relaxed);
        failures = totalFailures.load(std::memory_order_relaxed);
        
        leaks = 0;
        for (size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            if (trackingArena[i].load(std::memory_order_relaxed) != 0) {
                leaks++;
            }
        }
    }
    
    void reset() {
        totalAllocations.store(0);
        totalFrees.store(0);
        totalFailures.store(0);
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }
};
constexpr size_t ParallelTracker::MAX_TRACKED_OBJECTS;


// what each worker thread does
void workerThread(TestSlabArena& arena, ParallelTracker& tracker, 
                 uint16_t threadId, size_t iterations, 
                 std::atomic<bool>& shouldStop) {
    
    std::random_device rd;
    std::mt19937 gen(rd() ^ threadId);
    std::uniform_int_distribution<> sizeDist(8, 512);
    std::uniform_int_distribution<> actionDist(0, 100);
    std::uniform_int_distribution<> holdDist(0, 50);
    
    std::vector<std::pair<void*, size_t>> localAllocations;
    localAllocations.reserve(100);
    
    size_t localAllocs = 0, localFrees = 0;
    size_t localAllocatorFails = 0, localTrackerFails = 0;
    size_t localDataCorruption = 0, localFreeFails = 0;    

    for (size_t i = 0; i < iterations && !shouldStop.load(); i++) {
        int action = actionDist(gen);
        
        // 70% chance to alloc, 30% chance to free
        if (action < 70 || localAllocations.empty()) {
            // try alloc
            size_t objSize = sizeDist(gen);
            TestAllocator allocator(arena, objSize);
            
            void* ptr = allocator.alloc();
            if (ptr) {
                if (tracker.recordAllocation(ptr, objSize, threadId, arena)) {
                    localAllocations.push_back({ptr, objSize});
                    localAllocs++;
                    
                    // scribble some data so we can later check corruption
                    if (objSize >= 4) {
                        uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                        *intPtr = (threadId << 16) | (i & 0xFFFF);
                    }
                } else {
                    // tracker failed, free it back right away
                    TestAllocator freeAllocator(arena, objSize);
                    freeAllocator.free(ptr);
                    localTrackerFails++;
                }
            } else {
                localAllocatorFails++;
            }
        } else {
            // pick a random alloc and free it
            if (!localAllocations.empty()) {
                std::uniform_int_distribution<> indexDist(0, localAllocations.size() - 1);
                size_t index = indexDist(gen);
                
                void* ptr = localAllocations[index].first;
                size_t objSize = localAllocations[index].second;
                
                // sanity check data
                if (objSize >= 4) {
                    uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                    if ((*intPtr >> 16) != threadId) {
                        localDataCorruption++;
                        std::cout << "Thread " << threadId << ": data corruption detected!" << std::endl;
                    }
                }
                
                TestAllocator allocator(arena, objSize);
                if (allocator.free(ptr)) {
                    if (tracker.recordFree(ptr, threadId, arena)) {
                        localFrees++;
                    } else {
                        localTrackerFails++;
                    }
                } else {
                    localFreeFails++;
                }
                
                // pop it from local vector
                localAllocations[index] = localAllocations.back();
                localAllocations.pop_back();
            }
        }
        
        // sometimes just yield to mess with timing
        if (holdDist(gen) == 0) {
            std::this_thread::yield();
        }
    }
    
    // free whatever’s left
    for (const auto& alloc : localAllocations) {
        TestAllocator allocator(arena, alloc.second);
        if (allocator.free(alloc.first)) {
            tracker.recordFree(alloc.first, threadId, arena);
        }
    }
    
    std::cout << "Thread " << threadId << " done: " 
              << localAllocs << " allocs, " << localFrees << " frees, " 
              << localAllocatorFails << " alloc_fails, "
              << localTrackerFails << " tracker_fails, "
              << localDataCorruption << " corruptions, "
              << localFreeFails << " free_fails" << std::endl;
}

void testBasicParallel() {
    std::cout << "=== Basic Parallel Test ===" << std::endl;
    
    TestSlabArena arena;
    ParallelTracker tracker;
    
    const size_t numThreads = 4;
    const size_t iterationsPerThread = 1000;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up threads
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // wait for them
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    // print stats
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nResults:" << std::endl;
    std::cout << "Duration: " << duration.count() << "ms" << std::endl;
    std::cout << "Allocs: " << totalAllocs << std::endl;
    std::cout << "Frees: " << totalFrees << std::endl;
    std::cout << "Failures: " << totalFailures << std::endl;
    std::cout << "Leaks: " << totalLeaks << std::endl;
    std::cout << "Success rate: " << (100.0 * (totalAllocs + totalFrees)) / 
                                      (totalAllocs + totalFrees + totalFailures) << "%" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) no leaks" << std::endl;
    } else {
        std::cout << ":( leaks detected!" << std::endl;
    }
    
    std::cout << std::endl;
}

void testHighContentionParallel() {
    std::cout << "=== High contention parallel test ===" << std::endl;
    
    TestSlabArena arena;
    ParallelTracker tracker;
    
    const size_t numThreads = std::thread::hardware_concurrency() * 2;
    const size_t iterationsPerThread = 2000;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up more threads than cores
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // let it run a bit then stop
    std::this_thread::sleep_for(std::chrono::seconds(5));
    shouldStop.store(true);
    
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nresults:" << std::endl;
    std::cout << "threads: " << numThreads << std::endl;
    std::cout << "duration: " << duration.count() << "ms" << std::endl;
    std::cout << "allocs: " << totalAllocs << std::endl;
    std::cout << "frees: " << totalFrees << std::endl;
    std::cout << "failures: " << totalFailures << std::endl;
    std::cout << "leaks: " << totalLeaks << std::endl;
    std::cout << "throughput: " << (totalAllocs + totalFrees) * 1000 / duration.count() 
              << " ops/sec" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) No leaks under contention" << std::endl;
    } else {
        std::cout << ":( Leaks under contention!" << std::endl;
    }
    
    std::cout << std::endl;
}

void testStressTest() {
    std::cout << "=== Stress Test ===" << std::endl;
    
    TestSlabArena arena;
    ParallelTracker tracker;
    
    const size_t numThreads = 8;
    const size_t iterationsPerThread = 5000;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    // run a few rounds
    for (int round = 0; round < 3; round++) {
        std::cout << "Round " << (round + 1) << std::endl;
        tracker.reset();
        threads.clear();
        
        auto start = std::chrono::high_resolution_clock::now();
        
        for (size_t i = 0; i < numThreads; i++) {
            threads.emplace_back(workerThread, std::ref(arena), std::ref(tracker),
                                static_cast<uint16_t>(i + 1), iterationsPerThread, 
                                std::ref(shouldStop));
        }
        
        for (auto& thread : threads) {
            thread.join();
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        
        size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
        tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
        
        std::cout << "  Round " << (round + 1) << ": " << duration.count() << "ms, "
                  << totalAllocs << " allocs, " << totalFrees << " frees, "
                  << totalLeaks << " leaks" << std::endl;
    }
    
    std::cout << ":) Stress test done!" << std::endl << std::endl;
}

int main() {
    std::cout << "Parallel Test" << std::endl;
    std::cout << "==============" << std::endl << std::endl;
    
    std::cout << "Hardware concurrency: " << std::thread::hardware_concurrency() << " threads" << std::endl;
    std::cout << "Test Slab Arena Size: " << TestSlabArena::SLAB_COUNT << " slabs" << std::endl;
    std::cout << "Slab Size: " << TestSlabArena::slabType::SIZE << " bytes" << std::endl << std::endl;
    
    try {
        testBasicParallel();
        testHighContentionParallel();
        testStressTest();
        
        std::cout << "All tests done!" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return 1;
    } catch (...) {
        std::cerr << "Unknown exception!" << std::endl;
        return 1;
    }
    
    return 0;
}
