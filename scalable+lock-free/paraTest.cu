#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thread>
#include <atomic>
#include <random>
#include <chrono>
#include <cassert>
#include <unordered_set>
#include <mutex>

#include "log.h"
#include "allocator.h"



// global tracker for allocations/frees using an atomic bitmask
template <typename SIZE_TYPE>
class ParallelTracker {
public:
    static const size_t MAX_TRACKED_OBJECTS = SIZE_TYPE::VALUE;
    
private:
    // each entry: [31:16] = size, [15:0] = thread_id, 0 = free
    std::atomic<uint32_t> trackingArena[MAX_TRACKED_OBJECTS];
    std::atomic<size_t> currentByteTotal{0};
    std::atomic<size_t> minimumRefusalTotal{MAX_TRACKED_OBJECTS*8};

    std::atomic<size_t> totalAllocations{0};
    std::atomic<size_t> totalFrees{0};
    std::atomic<size_t> totalFailures{0};

    mutable std::mutex debugMutex;
    
public:

    ParallelTracker() {
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {

            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }


    size_t getCurrentByteTotal() {
        return currentByteTotal.load();
    }

    void logRefusalAtTotal(size_t total) {
        size_t expected = minimumRefusalTotal.load();
        while ( (expected > total) && (minimumRefusalTotal.compare_exchange_strong(expected,total,std::memory_order_acq_rel)) ) {}
    }
    

    // figure out arena index for a pointer
    std::pair<size_t, AllocationError> getIndexForPointer(void* ptr, TestSlabArena& arena) {
        if (!ptr) {
            return {MAX_TRACKED_OBJECTS, AllocationError::NULL_POINTER};
        }

        // which slab?
        auto slabIndex = arena.slabIndexFor(ptr);
        if (slabIndex >= TestSlabArena::SLAB_COUNT) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: ptr=" << ptr << " -> slabIndex=" << slabIndex 
                        << " (>= SLAB_COUNT=" << TestSlabArena::SLAB_COUNT << ")" << std::endl;
            
            // LSF analysis - show pointer details
            std::cout << "    LSF: ptr=0x" << std::hex << reinterpret_cast<uintptr_t>(ptr) 
                        << std::dec << " (LSF=" << (reinterpret_cast<uintptr_t>(ptr) & 0xF) << ")" << std::endl;
            std::cout << "    Checking all slabs for potential matches:" << std::endl;
            for (size_t i = 0; i < TestSlabArena::SLAB_COUNT; i++) {
                auto& slab = arena.slabAt(i);
                char* slabStart = reinterpret_cast<char*>(&slab);
                char* slabEnd = slabStart + TestSlabArena::slabType::SIZE;
                char* ptrChar = static_cast<char*>(ptr);
                
                if (ptrChar >= slabStart && ptrChar < slabEnd) {
                    std::cout << "      Found ptr in slab " << i << " (range: " 
                                << static_cast<void*>(slabStart) << " - " 
                                << static_cast<void*>(slabEnd) << ")" << std::endl;
                }
            }
            
            return {MAX_TRACKED_OBJECTS, AllocationError::INVALID_SLAB_INDEX};
        }

        // read proxy + slab
        auto& proxy = arena.proxyAt(slabIndex).data;
        if (proxy.getSize() == 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: slabIndex=" << slabIndex << " not bound (size=0)" << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::SLAB_NOT_BOUND};
        }

        auto& slab = arena.slabAt(slabIndex);

        // re-compute object index like proxy.free()
        char* p   = static_cast<char*>(ptr);
        char* sb  = reinterpret_cast<char*>(&slab);

        size_t objectSize   = proxy.getSize();
        size_t maxObjCount  = proxy.slabObjCount(objectSize);
        if (maxObjCount == 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: slabIndex=" << slabIndex << " objectSize=" << objectSize 
                        << " -> maxObjCount=0" << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::OBJECT_INDEX_OUT_OF_RANGE};
        }

        // number of mask elements and their byte size
        using allocMaskElem = typename TestSlabArena::slabProxyType::allocMaskElem;
        constexpr size_t ELEM_BITS = sizeof(allocMaskElem) * 8;

        size_t maskCount = (maxObjCount + ELEM_BITS - 1) / ELEM_BITS;
        size_t maskBytes = maskCount * sizeof(allocMaskElem);

        size_t firstObjOffset = maskBytes;

        size_t byteOffset = static_cast<size_t>(p - sb);
        if (byteOffset < firstObjOffset) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: byteOffset=" << byteOffset << " < firstObjOffset=" 
                        << firstObjOffset << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::OFFSET_TOO_SMALL};
        }

        size_t objOffset = byteOffset - firstObjOffset;
        if (objOffset % objectSize != 0) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: objOffset=" << objOffset << " % objectSize=" 
                        << objectSize << " = " << (objOffset % objectSize) << " (not aligned)" << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::MISALIGNED_OBJECT};
        }

        size_t objIndex = objOffset / objectSize;
        if (objIndex >= maxObjCount) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: objIndex=" << objIndex << " >= maxObjCount=" 
                        << maxObjCount << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::OBJECT_INDEX_OUT_OF_RANGE};
        }

        // flat index with fixed stride
        size_t idx = static_cast<size_t>(slabIndex) * PER_SLAB_STRIDE + objIndex;
        if (idx >= MAX_TRACKED_OBJECTS) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "    DEBUG: computed index=" << idx << " >= MAX_TRACKED_OBJECTS=" 
                        << MAX_TRACKED_OBJECTS << std::endl;
            return {MAX_TRACKED_OBJECTS, AllocationError::TRACKING_INDEX_OUT_OF_RANGE};
        }

        return {idx, AllocationError::SUCCESS};
    } // end of getInd
    
    // wrapper
    size_t getIndexForPtr(void* ptr, TestSlabArena& arena) {

        if (!ptr) return MAX_TRACKED_OBJECTS;
        
        char* ptrChar = static_cast<char*>(ptr);
        char* arenaBase = static_cast<char*>(static_cast<void*>(&arena));
        
        size_t offset = ptrChar - arenaBase;
        size_t index = offset / 8; // assuming 64-byte chunks
        
        return (index < MAX_TRACKED_OBJECTS) ? index : MAX_TRACKED_OBJECTS;

    }
    
    // log an allocation with CAS
    bool recordAllocation(void* ptr, size_t size, uint16_t threadId, TestSlabArena& arena) {
        size_t index = getIndexForPtr(ptr, arena);
        if (index >= MAX_TRACKED_OBJECTS) {
            Log() << "Failed: Allocation index for size "<< size << " at " << ptr <<" exceeds arena bounds!" << std::endl;

            return false;
        }
        
        uint32_t newValue = (static_cast<uint32_t>(size & 0xFFFF) << 16) | (threadId & 0xFFFF);
        uint32_t expected = 0;
        
        if (trackingArena[index].compare_exchange_strong(expected, newValue, std::memory_order_acq_rel)) {
            totalAllocations.fetch_add(1, std::memory_order_relaxed);
            currentByteTotal.fetch_add(size, std::memory_order_relaxed);
            return true;
        }
        uint32_t bad_size = (expected>>16) & 0xFFFF;
        uint32_t bad_id   = expected & 0xFFFF;
        Log() << "Failed to allocate size " << size << ". Expected 0, but found ("<<bad_size<<","<<bad_id<<")" << std::endl;

        totalFailures.fetch_add(1, std::memory_order_relaxed);
        return false;
    } // uped
    
    // log a free with CAS
    bool recordFree(void* ptr, uint16_t threadId, TestSlabArena& arena) {
        auto result = getIndexForPointer(ptr, arena);
        size_t index = result.first;
        AllocationError error = result.second;
        
        if (error != AllocationError::SUCCESS) {
            std::lock_guard<std::mutex> lock(debugMutex);
            std::cout << "FREE FAIL: Thread=" << threadId << " ptr=" << ptr 
                      << " ind=" << index << " error=" << errorToString(error) << std::endl;
            return false; // Don't count as total failure since this is just tracking
        }

        uint32_t curr = trackingArena[index].load(std::memory_order_acquire);
        while (curr != 0) {
            // clear to 0; CAS updates `cur` on failure
            if (trackingArena[index].compare_exchange_weak(curr, 0, std::memory_order_acq_rel)) {
                totalFrees.fetch_add(1, std::memory_order_relaxed);
                return true;
            }
        }
        
        if (trackingArena[index].compare_exchange_strong(current, 0, std::memory_order_acq_rel)) {
            totalFrees.fetch_add(1, std::memory_order_relaxed);
            uint32_t size = (current>>16) & 0xFFFF;
            currentByteTotal.fetch_sub(size, std::memory_order_relaxed);
            return true;
        }
        
        // Treat as success since the slot is clear
        return true;
    }
 
    
    // dump stats
    void getStats(size_t& allocs, size_t& frees, size_t& failures, size_t& leaks) {
        allocs = totalAllocations.load(std::memory_order_relaxed);
        frees = totalFrees.load(std::memory_order_relaxed);
        failures = totalFailures.load(std::memory_order_relaxed);
        
        leaks = 0;
        for (size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            if (trackingArena[i].load(std::memory_order_relaxed) != 0) {
                leaks++;
            }
        }
    }
    
    void reset() {
        totalAllocations.store(0);
        totalFrees.store(0);
        totalFailures.store(0);
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }
};
constexpr size_t ParallelTracker::MAX_TRACKED_OBJECTS;


// what each worker thread does
template<typename SIZE_TYPE>
void workerThread(TestSlabArena& arena, ParallelTracker<SIZE_TYPE>& tracker, 
                 uint16_t threadId, size_t iterations, 
                 std::atomic<bool>& shouldStop) {
    
    std::random_device rd;
    std::mt19937 gen(rd() ^ threadId);
    std::uniform_int_distribution<> sizeDist(3,9);
    std::uniform_int_distribution<> actionDist(0, 100);
    std::uniform_int_distribution<> holdDist(0, 50);
    
    std::vector<std::pair<void*, size_t>> localAllocations;
    localAllocations.reserve(100);
    
    size_t localAllocs = 0, localFrees = 0;
    size_t localAllocatorFails = 0, localTrackerFails = 0;
    size_t localDataCorruption = 0, localFreeFails = 0;    

    for (size_t i = 0; i < iterations && !shouldStop.load(); i++) {
        int action = actionDist(gen);
        
        // 70% chance to alloc, 30% chance to free
        if (action < 70 || localAllocations.empty()) {
            // try alloc
            size_t objSize = 1<<sizeDist(gen);
            TestAllocator allocator(arena, objSize);
            
            void* ptr = allocator.alloc();
            //Log() << "Allocated size " << objSize << " at " << ptr << std::endl;
            if (ptr) {
                if (tracker.recordAllocation(ptr, objSize, threadId, arena)) {
                    localAllocations.push_back({ptr, objSize});
                    localAllocs++;
                    
                    // scribble some data so we can later check corruption
                    if (objSize >= 4) {
                        uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                        *intPtr = (threadId << 16) | (i & 0xFFFF);
                    }
                } else {
                    Log() << "Failed to record allocation of size " << objSize << " at " << ptr << std::endl;
                    // tracker failed, free it back right away
                    TestAllocator freeAllocator(arena, objSize);
                    freeAllocator.free(ptr);
                    localTrackerFails++;
                }
            } else {
                size_t total = tracker.getCurrentByteTotal();
                tracker.logRefusalAtTotal(total);
                float proportion = ((float)total) / (ParallelTracker<SIZE_TYPE>::MAX_TRACKED_OBJECTS*8.0f);
                Log() << "Failed to  allocate object of size " << objSize << " with "
                      << (100.0*(1.0-proportion)) << "% capacity left"<<  std::endl;

            }
        } else {
            // pick a random alloc and free it
            if (!localAllocations.empty()) {
                std::uniform_int_distribution<> indexDist(0, localAllocations.size() - 1);
                size_t index = indexDist(gen);
                
                void* ptr = localAllocations[index].first;
                //Log() << "Deallocating at " << ptr << std::endl;
                size_t objSize = localAllocations[index].second;
                
                // sanity check data
                if (objSize >= 4) {
                    uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                    if ((*intPtr >> 16) != threadId) {
                        localDataCorruption++;
                        std::cout << "Thread " << threadId << ": data corruption detected!" << std::endl;
                    }
                }
                
                TestAllocator allocator(arena, objSize);
                if (allocator.free(ptr)) {
                    if (tracker.recordFree(ptr, threadId, arena)) {
                        localFrees++;
                    } else {
                        localErrors++;
                        Log() << "Failed to record deallocation of size " << objSize << " at " << ptr << std::endl;
                    }
                } else {
                    localErrors++;
                    Log() << "Failed to deallocate object of size " << objSize << " at " << ptr << std::endl;
                        //localTrackerFails++;
             
                }
                
                // pop it from local vector
                localAllocations[index] = localAllocations.back();
                localAllocations.pop_back();
            }
        }
        
        // sometimes just yield to mess with timing
        if (holdDist(gen) == 0) {
            std::this_thread::yield();
        }
    }
    
    // free whatever’s left
    for (const auto& alloc : localAllocations) {
        TestAllocator allocator(arena, alloc.second);
        if (allocator.free(alloc.first)) {
            tracker.recordFree(alloc.first, threadId, arena);
        }
    }
    
    std::cout << "Thread " << threadId << " done: " 
              << localAllocs << " allocs, " << localFrees << " frees, " 
              << localAllocatorFails << " alloc_fails, "
              << localTrackerFails << " tracker_fails, "
              << localDataCorruption << " corruptions, "
              << localFreeFails << " free_fails" << std::endl;
}

void testBasicParallel() {
    std::cout << "=== Basic Parallel Test ===" << std::endl;
    
    TestSlabArena *arena_ptr = new TestSlabArena;
    if (!arena_ptr) {
        return;
    }
    TestSlabArena &arena = *arena_ptr;
    constexpr size_t OBJECT_COUNT = TestSlabArena::SLAB_COUNT * TestSlabArena::slabType::SIZE / 8;
    typedef Size<OBJECT_COUNT> SizeType;
    ParallelTracker<SizeType> *tracker_ptr = new ParallelTracker<SizeType>;
    ParallelTracker<SizeType> &tracker = *tracker_ptr;
    
    const size_t numThreads = 2;
    const size_t iterationsPerThread = 1024;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up threads
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread<SizeType>, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // wait for them
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    // print stats
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nResults:" << std::endl;
    std::cout << "Duration: " << duration.count() << "ms" << std::endl;
    std::cout << "Allocs: " << totalAllocs << std::endl;
    std::cout << "Frees: " << totalFrees << std::endl;
    std::cout << "Failures: " << totalFailures << std::endl;
    std::cout << "Leaks: " << totalLeaks << std::endl;
    std::cout << "Success rate: " << (100.0 * (totalAllocs + totalFrees)) / 
                                      (totalAllocs + totalFrees + totalFailures) << "%" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) no leaks" << std::endl;
    } else {
        std::cout << ":( leaks detected!" << std::endl;
    }
    
    std::cout << std::endl;
    delete arena_ptr;
}

void testHighContentionParallel() {
    std::cout << "=== High contention parallel test ===" << std::endl;
    
    TestSlabArena arena;
    constexpr size_t OBJECT_COUNT = TestSlabArena::SLAB_COUNT * TestSlabArena::slabType::SIZE / 8;
    typedef Size<OBJECT_COUNT> SizeType;
    ParallelTracker<SizeType> tracker;
    
    const size_t numThreads = 1; std::thread::hardware_concurrency() * 2;
    const size_t iterationsPerThread = 1;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up more threads than cores
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread<SizeType>, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // let it run a bit then stop
    std::this_thread::sleep_for(std::chrono::seconds(5));
    shouldStop.store(true);
    
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nresults:" << std::endl;
    std::cout << "threads: " << numThreads << std::endl;
    std::cout << "duration: " << duration.count() << "ms" << std::endl;
    std::cout << "allocs: " << totalAllocs << std::endl;
    std::cout << "frees: " << totalFrees << std::endl;
    std::cout << "failures: " << totalFailures << std::endl;
    std::cout << "leaks: " << totalLeaks << std::endl;
    std::cout << "throughput: " << (totalAllocs + totalFrees) * 1000 / duration.count() 
              << " ops/sec" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) No leaks under contention" << std::endl;
    } else {
        std::cout << ":( Leaks under contention!" << std::endl;
    }
    
    std::cout << std::endl;
}


void testStressTest() {
    std::cout << "=== Stress Test ===" << std::endl;
    
    TestSlabArena arena;
    constexpr size_t OBJECT_COUNT = TestSlabArena::SLAB_COUNT * TestSlabArena::slabType::SIZE / 8;
    typedef Size<OBJECT_COUNT> SizeType;
    ParallelTracker<SizeType> tracker;
    
    const size_t numThreads = 1;
    const size_t iterationsPerThread = 1;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    // run a few rounds
    for (int round = 0; round < 1; round++) {
        std::cout << "Round " << (round + 1) << std::endl;
        tracker.reset();
        threads.clear();
        
        auto start = std::chrono::high_resolution_clock::now();
        
        for (size_t i = 0; i < numThreads; i++) {
            threads.emplace_back(workerThread<SizeType>, std::ref(arena), std::ref(tracker),
                                static_cast<uint16_t>(i + 1), iterationsPerThread, 
                                std::ref(shouldStop));
        }
        
        for (auto& thread : threads) {
            thread.join();
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        
        size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
        tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
        
        std::cout << "  Round " << (round + 1) << ": " << duration.count() << "ms, "
                  << totalAllocs << " allocs, " << totalFrees << " frees, "
                  << totalLeaks << " leaks" << std::endl;
    }
    
    std::cout << ":) Stress test done!" << std::endl << std::endl;
}

int main() {
    std::cout << "Parallel Test" << std::endl;
    std::cout << "==============" << std::endl << std::endl;
    
    std::cout << "Hardware concurrency: " << std::thread::hardware_concurrency() << " threads" << std::endl;
    std::cout << "Test Slab Arena Size: " << TestSlabArena::SLAB_COUNT << " slabs" << std::endl;
    std::cout << "Slab Size: " << TestSlabArena::slabType::SIZE << " bytes" << std::endl << std::endl;
    
    try {
        testBasicParallel();
        //testHighContentionParallel();
        //testStressTest();
        
        std::cout << "All tests done!" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return 1;
    } catch (...) {
        std::cerr << "Unknown exception!" << std::endl;
        return 1;
    }
    
    return 0;
}
