#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thread>
#include <atomic>
#include <random>
#include <chrono>
#include <cassert>
#include <unordered_set>
#include <mutex>

#include "log.h"
#include "allocator.h"


// global tracker for allocations/frees using an atomic bitmask
template <typename SIZE_TYPE>
class ParallelTracker {
public:
    static const size_t MAX_TRACKED_OBJECTS = SIZE_TYPE::VALUE;
    
private:
    // each entry: [31:16] = size, [15:0] = thread_id, 0 = free
    std::atomic<uint32_t> trackingArena[MAX_TRACKED_OBJECTS];
    std::atomic<size_t> currentByteTotal{0};
    std::atomic<size_t> minimumRefusalTotal{MAX_TRACKED_OBJECTS*8};
    std::atomic<size_t> totalAllocations{0};
    std::atomic<size_t> totalFrees{0};
    std::atomic<size_t> totalFailures{0};
    
public:

    ParallelTracker() {
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }

    size_t getCurrentByteTotal() {
        return currentByteTotal.load();
    }

    void logRefusalAtTotal(size_t total) {
        size_t expected = minimumRefusalTotal.load();
        while ( (expected > total) && (minimumRefusalTotal.compare_exchange_strong(expected,total,std::memory_order_acq_rel)) ) {}
    }
    
    // figure out arena index for a pointer
    size_t getIndexForPtr(void* ptr, TestSlabArena& arena) {
        if (!ptr) return MAX_TRACKED_OBJECTS;
        
        char* ptrChar = static_cast<char*>(ptr);
        char* arenaBase = static_cast<char*>(static_cast<void*>(&arena));
        
        size_t offset = ptrChar - arenaBase;
        size_t index = offset / 8; // assuming 64-byte chunks
        
        return (index < MAX_TRACKED_OBJECTS) ? index : MAX_TRACKED_OBJECTS;
    }
    
    // log an allocation with CAS
    bool recordAllocation(void* ptr, size_t size, uint16_t threadId, TestSlabArena& arena) {
        size_t index = getIndexForPtr(ptr, arena);
        if (index >= MAX_TRACKED_OBJECTS) {
            Log() << "Failed: Allocation index for size "<< size << " at " << ptr <<" exceeds arena bounds!" << std::endl;
            return false;
        }
        
        uint32_t newValue = (static_cast<uint32_t>(size & 0xFFFF) << 16) | (threadId & 0xFFFF);
        uint32_t expected = 0;
        
        if (trackingArena[index].compare_exchange_strong(expected, newValue, std::memory_order_acq_rel)) {
            totalAllocations.fetch_add(1, std::memory_order_relaxed);
            currentByteTotal.fetch_add(size, std::memory_order_relaxed);
            return true;
        }
        uint32_t bad_size = (expected>>16) & 0xFFFF;
        uint32_t bad_id   = expected & 0xFFFF;
        Log() << "Failed to allocate size " << size << ". Expected 0, but found ("<<bad_size<<","<<bad_id<<")" << std::endl;
        totalFailures.fetch_add(1, std::memory_order_relaxed);
        return false;
    }
    
    // log a free with CAS
    bool recordFree(void* ptr, uint16_t threadId, TestSlabArena& arena) {
        size_t index = getIndexForPtr(ptr, arena);
        if (index >= MAX_TRACKED_OBJECTS) return false;
        
        uint32_t current = trackingArena[index].load(std::memory_order_acquire);
        
        // check if it was really allocated by this thread
        if ((current & 0xFFFF) != threadId || current == 0) {
            totalFailures.fetch_add(1, std::memory_order_relaxed);
            return false;
        }
        
        if (trackingArena[index].compare_exchange_strong(current, 0, std::memory_order_acq_rel)) {
            totalFrees.fetch_add(1, std::memory_order_relaxed);
            uint32_t size = (current>>16) & 0xFFFF;
            currentByteTotal.fetch_sub(size, std::memory_order_relaxed);
            return true;
        }
        
        totalFailures.fetch_add(1, std::memory_order_relaxed);
        return false;
    }
    
    // dump stats
    void getStats(size_t& allocs, size_t& frees, size_t& failures, size_t& leaks) {
        allocs = totalAllocations.load(std::memory_order_relaxed);
        frees = totalFrees.load(std::memory_order_relaxed);
        failures = totalFailures.load(std::memory_order_relaxed);
        
        leaks = 0;
        for (size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            if (trackingArena[i].load(std::memory_order_relaxed) != 0) {
                leaks++;
            }
        }
    }
    
    void reset() {
        totalAllocations.store(0);
        totalFrees.store(0);
        totalFailures.store(0);
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }
};

// what each worker thread does
template<typename SIZE_TYPE>
void workerThread(TestSlabArena& arena, ParallelTracker<SIZE_TYPE>& tracker, 
                 uint16_t threadId, size_t iterations, 
                 std::atomic<bool>& shouldStop) {
    
    std::random_device rd;
    std::mt19937 gen(rd() ^ threadId);
    std::uniform_int_distribution<> sizeDist(3,9);
    std::uniform_int_distribution<> actionDist(0, 100);
    std::uniform_int_distribution<> holdDist(0, 50);
    
    std::vector<std::pair<void*, size_t>> localAllocations;
    localAllocations.reserve(100);
    
    size_t localAllocs = 0, localFrees = 0, localErrors = 0;
    
    for (size_t i = 0; i < iterations && !shouldStop.load(); i++) {
        int action = actionDist(gen);
        
        // 70% chance to alloc, 30% chance to free
        if (action < 70 || localAllocations.empty()) {
            // try alloc
            size_t objSize = 1<<sizeDist(gen);
            TestAllocator allocator(arena, objSize);
            
            void* ptr = allocator.alloc();
            //Log() << "Allocated size " << objSize << " at " << ptr << std::endl;
            if (ptr) {
                if (tracker.recordAllocation(ptr, objSize, threadId, arena)) {
                    localAllocations.push_back({ptr, objSize});
                    localAllocs++;
                    
                    // scribble some data so we can later check corruption
                    if (objSize >= 4) {
                        uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                        *intPtr = (threadId << 16) | (i & 0xFFFF);
                    }
                } else {
                    Log() << "Failed to record allocation of size " << objSize << " at " << ptr << std::endl;
                    // tracker failed, free it back right away
                    TestAllocator freeAllocator(arena, objSize);
                    freeAllocator.free(ptr);
                    localErrors++;
                }
            } else {
                size_t total = tracker.getCurrentByteTotal();
                tracker.logRefusalAtTotal(total);
                float proportion = ((float)total) / (ParallelTracker<SIZE_TYPE>::MAX_TRACKED_OBJECTS*8.0f);
                Log() << "Failed to  allocate object of size " << objSize << " with "
                      << (100.0*(1.0-proportion)) << "% capacity left"<<  std::endl;
            }
        } else {
            // pick a random alloc and free it
            if (!localAllocations.empty()) {
                std::uniform_int_distribution<> indexDist(0, localAllocations.size() - 1);
                size_t index = indexDist(gen);
                
                void* ptr = localAllocations[index].first;
                //Log() << "Deallocating at " << ptr << std::endl;
                size_t objSize = localAllocations[index].second;
                
                // sanity check data
                if (objSize >= 4) {
                    uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                    if ((*intPtr >> 16) != threadId) {
                        localErrors++;
                        std::cout << "Thread " << threadId << ": data corruption detected!" << std::endl;
                    }
                }
                
                TestAllocator allocator(arena, objSize);
                if (allocator.free(ptr)) {
                    if (tracker.recordFree(ptr, threadId, arena)) {
                        localFrees++;
                    } else {
                        localErrors++;
                        Log() << "Failed to record deallocation of size " << objSize << " at " << ptr << std::endl;
                    }
                } else {
                    localErrors++;
                    Log() << "Failed to deallocate object of size " << objSize << " at " << ptr << std::endl;
                }
                
                // pop it from local vector
                localAllocations[index] = localAllocations.back();
                localAllocations.pop_back();
            }
        }
        
        // sometimes just yield to mess with timing
        if (holdDist(gen) == 0) {
            std::this_thread::yield();
        }
    }
    
    // free whatever’s left
    for (const auto& alloc : localAllocations) {
        TestAllocator allocator(arena, alloc.second);
        if (allocator.free(alloc.first)) {
            tracker.recordFree(alloc.first, threadId, arena);
        }
    }
    
    std::cout << "Thread " << threadId << " done: " 
              << localAllocs << " allocs, " << localFrees << " frees, " 
              << localErrors << " errors" << std::endl;
}

void testBasicParallel() {
    std::cout << "=== Basic Parallel Test ===" << std::endl;
    
    TestSlabArena *arena_ptr = new TestSlabArena;
    if (!arena_ptr) {
        return;
    }
    TestSlabArena &arena = *arena_ptr;
    constexpr size_t OBJECT_COUNT = TestSlabArena::SLAB_COUNT * TestSlabArena::slabType::SIZE / 8;
    typedef Size<OBJECT_COUNT> SizeType;
    ParallelTracker<SizeType> *tracker_ptr = new ParallelTracker<SizeType>;
    ParallelTracker<SizeType> &tracker = *tracker_ptr;
    
    const size_t numThreads = 2;
    const size_t iterationsPerThread = 1024;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up threads
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread<SizeType>, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // wait for them
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    // print stats
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nResults:" << std::endl;
    std::cout << "Duration: " << duration.count() << "ms" << std::endl;
    std::cout << "Allocs: " << totalAllocs << std::endl;
    std::cout << "Frees: " << totalFrees << std::endl;
    std::cout << "Failures: " << totalFailures << std::endl;
    std::cout << "Leaks: " << totalLeaks << std::endl;
    std::cout << "Success rate: " << (100.0 * (totalAllocs + totalFrees)) / 
                                      (totalAllocs + totalFrees + totalFailures) << "%" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) no leaks" << std::endl;
    } else {
        std::cout << ":( leaks detected!" << std::endl;
    }
    
    std::cout << std::endl;
    delete arena_ptr;
}

void testHighContentionParallel() {
    std::cout << "=== high contention parallel test ===" << std::endl;
    
    TestSlabArena arena;
    constexpr size_t OBJECT_COUNT = TestSlabArena::SLAB_COUNT * TestSlabArena::slabType::SIZE / 8;
    typedef Size<OBJECT_COUNT> SizeType;
    ParallelTracker<SizeType> tracker;
    
    const size_t numThreads = 1; std::thread::hardware_concurrency() * 2;
    const size_t iterationsPerThread = 1;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up more threads than cores
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread<SizeType>, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // let it run a bit then stop
    std::this_thread::sleep_for(std::chrono::seconds(5));
    shouldStop.store(true);
    
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nresults:" << std::endl;
    std::cout << "threads: " << numThreads << std::endl;
    std::cout << "duration: " << duration.count() << "ms" << std::endl;
    std::cout << "allocs: " << totalAllocs << std::endl;
    std::cout << "frees: " << totalFrees << std::endl;
    std::cout << "failures: " << totalFailures << std::endl;
    std::cout << "leaks: " << totalLeaks << std::endl;
    std::cout << "throughput: " << (totalAllocs + totalFrees) * 1000 / duration.count() 
              << " ops/sec" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) No leaks under contention" << std::endl;
    } else {
        std::cout << ":( Leaks under contention!" << std::endl;
    }
    
    std::cout << std::endl;
}


void testStressTest() {
    std::cout << "=== Stress Test ===" << std::endl;
    
    TestSlabArena arena;
    constexpr size_t OBJECT_COUNT = TestSlabArena::SLAB_COUNT * TestSlabArena::slabType::SIZE / 8;
    typedef Size<OBJECT_COUNT> SizeType;
    ParallelTracker<SizeType> tracker;
    
    const size_t numThreads = 1;
    const size_t iterationsPerThread = 1;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    // run a few rounds
    for (int round = 0; round < 1; round++) {
        std::cout << "Round " << (round + 1) << std::endl;
        tracker.reset();
        threads.clear();
        
        auto start = std::chrono::high_resolution_clock::now();
        
        for (size_t i = 0; i < numThreads; i++) {
            threads.emplace_back(workerThread<SizeType>, std::ref(arena), std::ref(tracker),
                                static_cast<uint16_t>(i + 1), iterationsPerThread, 
                                std::ref(shouldStop));
        }
        
        for (auto& thread : threads) {
            thread.join();
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        
        size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
        tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
        
        std::cout << "  Round " << (round + 1) << ": " << duration.count() << "ms, "
                  << totalAllocs << " allocs, " << totalFrees << " frees, "
                  << totalLeaks << " leaks" << std::endl;
    }
    
    std::cout << ":) Stress test done!" << std::endl << std::endl;
}

int main() {
    std::cout << "Parallel Test" << std::endl;
    std::cout << "==============" << std::endl << std::endl;
    
    std::cout << "Hardware concurrency: " << std::thread::hardware_concurrency() << " threads" << std::endl;
    std::cout << "Test Slab Arena Size: " << TestSlabArena::SLAB_COUNT << " slabs" << std::endl;
    std::cout << "Slab Size: " << TestSlabArena::slabType::SIZE << " bytes" << std::endl << std::endl;
    
    try {
        testBasicParallel();
        //testHighContentionParallel();
        //testStressTest();
        
        std::cout << "All tests done!" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return 1;
    } catch (...) {
        std::cerr << "Unknown exception!" << std::endl;
        return 1;
    }
    
    return 0;
}
