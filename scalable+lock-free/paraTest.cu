#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thread>
#include <atomic>
#include <random>
#include <chrono>
#include <cassert>
#include <unordered_set>
#include <mutex>

#include "allocator.h"

// global tracker for allocations/frees using an atomic bitmask
class ParallelTracker {
public:
    static const size_t MAX_TRACKED_OBJECTS = 100000;
    
private:
    // each entry: [31:16] = size, [15:0] = thread_id, 0 = free
    std::atomic<uint32_t> trackingArena[MAX_TRACKED_OBJECTS];
    std::atomic<size_t> totalAllocations{0};
    std::atomic<size_t> totalFrees{0};
    std::atomic<size_t> totalFailures{0};
    
public:
    ParallelTracker() {
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }
    
    // figure out arena index for a pointer
    size_t getIndexForPtr(void* ptr, TestSlabArena& arena) {
        if (!ptr) return MAX_TRACKED_OBJECTS;
        
        char* ptrChar = static_cast<char*>(ptr);
        char* arenaBase = static_cast<char*>(static_cast<void*>(&arena));
        
        size_t offset = ptrChar - arenaBase;
        size_t index = offset / 64; // assuming 64-byte chunks
        
        return (index < MAX_TRACKED_OBJECTS) ? index : MAX_TRACKED_OBJECTS;
    }
    
    // log an allocation with CAS
    bool recordAllocation(void* ptr, size_t size, uint16_t threadId, TestSlabArena& arena) {
        size_t index = getIndexForPtr(ptr, arena);
        if (index >= MAX_TRACKED_OBJECTS) return false;
        
        uint32_t newValue = (static_cast<uint32_t>(size & 0xFFFF) << 16) | (threadId & 0xFFFF);
        uint32_t expected = 0;
        
        if (trackingArena[index].compare_exchange_strong(expected, newValue, std::memory_order_acq_rel)) {
            totalAllocations.fetch_add(1, std::memory_order_relaxed);
            return true;
        }
        
        totalFailures.fetch_add(1, std::memory_order_relaxed);
        return false;
    }
    
    // log a free with CAS
    bool recordFree(void* ptr, uint16_t threadId, TestSlabArena& arena) {
        size_t index = getIndexForPtr(ptr, arena);
        if (index >= MAX_TRACKED_OBJECTS) return false;
        
        uint32_t current = trackingArena[index].load(std::memory_order_acquire);
        
        // check if it was really allocated by this thread
        if ((current & 0xFFFF) != threadId || current == 0) {
            totalFailures.fetch_add(1, std::memory_order_relaxed);
            return false;
        }
        
        if (trackingArena[index].compare_exchange_strong(current, 0, std::memory_order_acq_rel)) {
            totalFrees.fetch_add(1, std::memory_order_relaxed);
            return true;
        }
        
        totalFailures.fetch_add(1, std::memory_order_relaxed);
        return false;
    }
    
    // dump stats
    void getStats(size_t& allocs, size_t& frees, size_t& failures, size_t& leaks) {
        allocs = totalAllocations.load(std::memory_order_relaxed);
        frees = totalFrees.load(std::memory_order_relaxed);
        failures = totalFailures.load(std::memory_order_relaxed);
        
        leaks = 0;
        for (size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            if (trackingArena[i].load(std::memory_order_relaxed) != 0) {
                leaks++;
            }
        }
    }
    
    void reset() {
        totalAllocations.store(0);
        totalFrees.store(0);
        totalFailures.store(0);
        for(size_t i = 0; i < MAX_TRACKED_OBJECTS; i++) {
            trackingArena[i].store(0, std::memory_order_relaxed);
        }
    }
};

// what each worker thread does
void workerThread(TestSlabArena& arena, ParallelTracker& tracker, 
                 uint16_t threadId, size_t iterations, 
                 std::atomic<bool>& shouldStop) {
    
    std::random_device rd;
    std::mt19937 gen(rd() ^ threadId);
    std::uniform_int_distribution<> sizeDist(8, 512);
    std::uniform_int_distribution<> actionDist(0, 100);
    std::uniform_int_distribution<> holdDist(0, 50);
    
    std::vector<std::pair<void*, size_t>> localAllocations;
    localAllocations.reserve(100);
    
    size_t localAllocs = 0, localFrees = 0, localErrors = 0;
    
    for (size_t i = 0; i < iterations && !shouldStop.load(); i++) {
        int action = actionDist(gen);
        
        // 70% chance to alloc, 30% chance to free
        if (action < 70 || localAllocations.empty()) {
            // try alloc
            size_t objSize = sizeDist(gen);
            TestAllocator allocator(arena, objSize);
            
            void* ptr = allocator.alloc();
            if (ptr) {
                if (tracker.recordAllocation(ptr, objSize, threadId, arena)) {
                    localAllocations.push_back({ptr, objSize});
                    localAllocs++;
                    
                    // scribble some data so we can later check corruption
                    if (objSize >= 4) {
                        uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                        *intPtr = (threadId << 16) | (i & 0xFFFF);
                    }
                } else {
                    // tracker failed, free it back right away
                    TestAllocator freeAllocator(arena, objSize);
                    freeAllocator.free(ptr);
                    localErrors++;
                }
            }
        } else {
            // pick a random alloc and free it
            if (!localAllocations.empty()) {
                std::uniform_int_distribution<> indexDist(0, localAllocations.size() - 1);
                size_t index = indexDist(gen);
                
                void* ptr = localAllocations[index].first;
                size_t objSize = localAllocations[index].second;
                
                // sanity check data
                if (objSize >= 4) {
                    uint32_t* intPtr = static_cast<uint32_t*>(ptr);
                    if ((*intPtr >> 16) != threadId) {
                        localErrors++;
                        std::cout << "Thread " << threadId << ": data corruption detected!" << std::endl;
                    }
                }
                
                TestAllocator allocator(arena, objSize);
                if (allocator.free(ptr)) {
                    if (tracker.recordFree(ptr, threadId, arena)) {
                        localFrees++;
                    } else {
                        localErrors++;
                    }
                } else {
                    localErrors++;
                }
                
                // pop it from local vector
                localAllocations[index] = localAllocations.back();
                localAllocations.pop_back();
            }
        }
        
        // sometimes just yield to mess with timing
        if (holdDist(gen) == 0) {
            std::this_thread::yield();
        }
    }
    
    // free whatever’s left
    for (const auto& alloc : localAllocations) {
        TestAllocator allocator(arena, alloc.second);
        if (allocator.free(alloc.first)) {
            tracker.recordFree(alloc.first, threadId, arena);
        }
    }
    
    std::cout << "Thread " << threadId << " done: " 
              << localAllocs << " allocs, " << localFrees << " frees, " 
              << localErrors << " errors" << std::endl;
}

void testBasicParallel() {
    std::cout << "=== Basic Parallel Test ===" << std::endl;
    
    TestSlabArena arena;
    ParallelTracker tracker;
    
    const size_t numThreads = 4;
    const size_t iterationsPerThread = 1000;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up threads
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // wait for them
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    // print stats
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nResults:" << std::endl;
    std::cout << "Duration: " << duration.count() << "ms" << std::endl;
    std::cout << "Allocs: " << totalAllocs << std::endl;
    std::cout << "Frees: " << totalFrees << std::endl;
    std::cout << "Failures: " << totalFailures << std::endl;
    std::cout << "Leaks: " << totalLeaks << std::endl;
    std::cout << "Success rate: " << (100.0 * (totalAllocs + totalFrees)) / 
                                      (totalAllocs + totalFrees + totalFailures) << "%" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) no leaks" << std::endl;
    } else {
        std::cout << ":( leaks detected!" << std::endl;
    }
    
    std::cout << std::endl;
}

void testHighContentionParallel() {
    std::cout << "=== high contention parallel test ===" << std::endl;
    
    TestSlabArena arena;
    ParallelTracker tracker;
    
    const size_t numThreads = std::thread::hardware_concurrency() * 2;
    const size_t iterationsPerThread = 2000;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // spin up more threads than cores
    for (size_t i = 0; i < numThreads; i++) {
        threads.emplace_back(workerThread, std::ref(arena), std::ref(tracker),
                            static_cast<uint16_t>(i + 1), iterationsPerThread, 
                            std::ref(shouldStop));
    }
    
    // let it run a bit then stop
    std::this_thread::sleep_for(std::chrono::seconds(5));
    shouldStop.store(true);
    
    for (auto& thread : threads) {
        thread.join();
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
    tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
    
    std::cout << "\nresults:" << std::endl;
    std::cout << "threads: " << numThreads << std::endl;
    std::cout << "duration: " << duration.count() << "ms" << std::endl;
    std::cout << "allocs: " << totalAllocs << std::endl;
    std::cout << "frees: " << totalFrees << std::endl;
    std::cout << "failures: " << totalFailures << std::endl;
    std::cout << "leaks: " << totalLeaks << std::endl;
    std::cout << "throughput: " << (totalAllocs + totalFrees) * 1000 / duration.count() 
              << " ops/sec" << std::endl;
    
    if (totalLeaks == 0) {
        std::cout << ":) No leaks under contention" << std::endl;
    } else {
        std::cout << ":( Leaks under contention!" << std::endl;
    }
    
    std::cout << std::endl;
}

void testStressTest() {
    std::cout << "=== Stress Test ===" << std::endl;
    
    TestSlabArena arena;
    ParallelTracker tracker;
    
    const size_t numThreads = 8;
    const size_t iterationsPerThread = 5000;
    
    std::atomic<bool> shouldStop{false};
    std::vector<std::thread> threads;
    
    // run a few rounds
    for (int round = 0; round < 3; round++) {
        std::cout << "Round " << (round + 1) << std::endl;
        tracker.reset();
        threads.clear();
        
        auto start = std::chrono::high_resolution_clock::now();
        
        for (size_t i = 0; i < numThreads; i++) {
            threads.emplace_back(workerThread, std::ref(arena), std::ref(tracker),
                                static_cast<uint16_t>(i + 1), iterationsPerThread, 
                                std::ref(shouldStop));
        }
        
        for (auto& thread : threads) {
            thread.join();
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        
        size_t totalAllocs, totalFrees, totalFailures, totalLeaks;
        tracker.getStats(totalAllocs, totalFrees, totalFailures, totalLeaks);
        
        std::cout << "  Round " << (round + 1) << ": " << duration.count() << "ms, "
                  << totalAllocs << " allocs, " << totalFrees << " frees, "
                  << totalLeaks << " leaks" << std::endl;
    }
    
    std::cout << ":) Stress test done!" << std::endl << std::endl;
}

int main() {
    std::cout << "Parallel Test" << std::endl;
    std::cout << "==============" << std::endl << std::endl;
    
    std::cout << "Hardware concurrency: " << std::thread::hardware_concurrency() << " threads" << std::endl;
    std::cout << "Test Slab Arena Size: " << TestSlabArena::SLAB_COUNT << " slabs" << std::endl;
    std::cout << "Slab Size: " << TestSlabArena::slabType::SIZE << " bytes" << std::endl << std::endl;
    
    try {
        testBasicParallel();
        testHighContentionParallel();
        testStressTest();
        
        std::cout << "All tests done!" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return 1;
    } catch (...) {
        std::cerr << "Unknown exception!" << std::endl;
        return 1;
    }
    
    return 0;
}
